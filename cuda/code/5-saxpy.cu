
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void saxpy(int32_t n, float alpha, const float *x, float *y) {
  int32_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid >= n)
    return;

  y[tid] += alpha * x[tid];
}

int main() {
  std::vector<float> x{1, 2, 3};
  float alpha = 2;
  std::vector<float> y{10, 20, 30};

  float *d_x;
  hipMalloc(&d_x, x.size() * sizeof(float));
  hipMemcpy(d_x, x.data(), x.size() * sizeof(float), hipMemcpyHostToDevice);

  float *d_y;
  hipMalloc(&d_y, x.size() * sizeof(float));
  hipMemcpy(d_y, y.data(), y.size() * sizeof(float), hipMemcpyHostToDevice);

  int32_t threads_per_block = 256;
  int32_t num_blokcs = (x.size() + threads_per_block - 1) / threads_per_block;
  saxpy<<<num_blokcs, threads_per_block>>>(x.size(), alpha, d_x, d_y);
  hipMemcpy(y.data(), d_y, y.size() * sizeof(float), hipMemcpyDeviceToHost);

  for (auto i : y)
    std::cout << i << "\n";
}
